/* 
 * Example of using reducing (tree) type algorithms to parallelize finding the minimum of
 * a set of numbers. On a GF 8600 GT the two parallel algorithms (sumControl = 0 or 1) 
 * are about 35 times faster than the serial algorithm also running on the GPU but using 
 * global memory (sumControl=2), for an array of 512 floats.  This is both because
 * the parallel algorithms scale as ln N while the serial algorithm scales as N, and because the
 * parallel algorithms use the shared memory while the serial one uses the (generally slower)
 * global memory in these tests. If we do the serial algorithm on the same computer but use
 * shared memory (set sumControl=3), the parallel algorithms are only 4.5 times faster.  Thus a
 * factor of around 10 in the speed up is because of using the shared memory in this example. Note, 
 * however, that shared memory can only be shared among the threads within a single block.
 * 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKSIZE 512 

// Define some GPU timing utilities. These are invoked from the host program. Usage:
//     START_GPU;
//         kernelFunction <<< numBlocks, threadsPerBlock >>> (args)
//     STOP_GPU;
//     PRINT_GPU
// in the host code. This estimates the time for the kernel kernelFunction to run on the GPU.
// For a more extensive discusion, see Section 5.1.2 of the CUDA Best Practices Guide at
// http://developer.download.nvidia.com/compute/DevZone/docs/html/C/doc/CUDA_C_Best_Practices_Guide.pdf

float timeGPU;
hipEvent_t start, stop;
#define START_GPU hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
#define STOP_GPU hipEventRecord(stop, 0); hipEventSynchronize(stop);\
   hipEventElapsedTime(&timeGPU, start, stop);\
   hipEventDestroy(start);hipEventDestroy(stop);
#define PRINT_GPU printf("\n\nTime to compute on GPU: %f ms \n", timeGPU);

// Define a utility to check for CUDA errors.  Place it immediately after a CUDA kernel
// call in the host code. The initial cudaDeviceSynchronize() command ensures that the device
// has completed all preceding requested tasks.

#define CUDA_ERROR_CHECK hipDeviceSynchronize(); hipError_t error = hipGetLastError();\
   if(error != hipSuccess){printf("***CUDA error: %s\n", hipGetErrorString(error)); exit(-1);}\
   else{printf("\nNo CUDA errors detected\n" );}
   

// Device code.  Examines the elements of the array Array and puts the min value in MinVal

__global__ void MinValKernel(float* Array, float* MinVal, int arraySize)
{
	__device__ float reductionMinVal(int, float *);

	*MinVal = reductionMinVal(arraySize, Array);
}

/*
  Function to do parallel reduction minVal. This should scale as ln N.  The parallel butterfly
  algorithm taken from the literature works generally.  My homegrown parallel version
  works as written for even number of entries in the array, so this algorithm can be used 
  for an odd number by padding the array with an added zero entry. Note that this version 
  assumes that all minVals are within one block, so a max of 512 threads on 1.1 devices
  (presently blocksize is set to 512).  One option for larger minVals is to break the array up 
  onto multiple blocks, use this algorithm on each block to get a block minVal, and then minVal
  the block minVals.
*/

__device__ float reductionMinVal(int length, float *array)
{
	float minVal;
	
	// = 0 or 1 for parallel with shared memory, 2 for serial with global, 3 for serial with shared
	int minValControl = 3;
	
	// Copy the array to be minValed into shared memory and initialize
	__shared__ float sarray[BLOCKSIZE];
	int i = threadIdx.x;
	sarray[i] = 0.0f;
	if(i<length) sarray[i] = array[i];
	__syncthreads();
	
	
	if(minValControl == 0)
	{
		// Parallel butterfly minVal
		// see http://cs.anu.edu.au/files/systems/GPUWksp/PDFs/02_CUDAParallelProgrammingModel.pdf
		
		for(int bit=BLOCKSIZE/2; bit>0; bit /= 2)
		{
			if(i<length)
			{
				float t = sarray[i];
				if(sarray[i^bit] < t) t = sarray[i^bit];
				__syncthreads();
				sarray[i] = t;
				__syncthreads();
			}
		}
		
		minVal = sarray[0];   // Contains the min value
	
	}
	else if(minValControl == 1)
	{
		// Another home-made parallel version of a reduction minVal. As written, this requires an even
		// number of entries in the array to be minValed, so pad with a zero to handle odd number
		// (or rewrite to deal with odd number).
		
		int L=length;
		int steps = 0;
		int inc = 1;
		float t = 0;
		
		while(L > 2 )
		{
			steps ++;
			if(i < (length-inc))
				t = sarray[i];
				if(sarray[i+inc] < t) t = sarray[i+inc];
				__syncthreads();
				sarray[i] = t;
				__syncthreads();
			inc *= 2;
			L /= 2;
		}
		
		minVal = sarray[0];   // This contains the minVal
	}
	else if(minValControl == 2)
	{	
	// For reference: Serial version of minimum finder running on the GPU and accessing global 
	// (not shared) memory.
	
		minVal = 1.0e+20;
		for(int i=0; i<length; i++)
		{
			if(array[i] < minVal)
			{
				minVal = array[i];
			}
		}
	}
	else
	{
		// For reference: Serial version of minimum finder running on the GPU and accessing shared 
		// memory.
		
		minVal = 1.0e+20;
		for(int i=0; i<length; i++)
		{
			if(sarray[i] < minVal)
			{
				minVal = sarray[i];
			}
		}
	}
	
	return minVal;
}



// Host code running on CPU
int main(void)
{
	int arraySize = 512;
	float* Array;
	float MinVal;
	
	Array = (float*) malloc(sizeof(float) * arraySize);
	
	// Fill array with some numbers.  Make it a monotonic function with minimum at
	// i = 300.  Then add some negative spikes to it so there are multiple local
	// minima.
	
	for(int i=0; i<arraySize; i++)
	{
		Array[i] = 0.005f*(float)(i-200)*(i-200) +13.0f;
	}
	
	// Add some negative spikes so that min is at i=100, corresponding to a value -39.6.
	Array[100] = -39.6f;
	Array[150] = 15.0f;
	Array[400] = 26.8f;
	Array[450] = -30.0f;
	
	// Set up device pointers
	float *devPtrArray;
	float* devPtrMinVal; 
	
	// Allocate device memory
	hipMalloc((void**)&devPtrMinVal, sizeof(float));
	hipMalloc((void**)&devPtrArray, arraySize * sizeof(float));
	
	// Copy array to device
	hipMemcpy(devPtrArray, Array, arraySize * sizeof(float), hipMemcpyHostToDevice);
	
	// Launch the kernel.
	
	START_GPU;     // Start timer for device code
	
	MinValKernel<<<1, 512>>>(devPtrArray, devPtrMinVal, arraySize);
	
	STOP_GPU;           // Stop timer for device code
	PRINT_GPU;          // Print timing for device code
	CUDA_ERROR_CHECK
	
	// Copy the minVal back from the GPU to the host
	
	hipMemcpy(&MinVal, devPtrMinVal, sizeof(float), hipMemcpyDeviceToHost);
	
	// Print out all the values in the array
	for(int i=0; i< arraySize; i++)
	{
		printf("%d %8.3e\n", i, Array[i]);
	}
	
	printf("\nTime to compute on GPU: %f ms \n", timeGPU);
	
	printf("\nMin value in array (GPU) = %6.2f\n\n", MinVal);
	 
    // Free the memory allocated on the device   
	hipFree(devPtrMinVal);
	hipFree(devPtrArray);
	
	// Free the memory allocated on the CPU
	free(Array);
	
    return 0;
	
}
