/* 
 * Example of using reducing (tree) type algorithms to parallelize finding the maximum of
 * a set of numbers. On a GF 8600 GT the two parallel algorithms (sumControl = 0 or 1) 
 * are about 35 times faster than the serial algorithm also running on the GPU but using 
 * global memory (sumControl=2), for an array of 512 floats.  This is both because
 * the parallel algorithms scale as ln N while the serial algorithm scales as N, and because the
 * parallel algorithms use the shared memory while the serial one uses the (generally slower)
 * global memory in these tests. If we do the serial algorithm on the same computer but use
 * shared memory (set sumControl=3), the parallel algorithms are only 4.5 times faster.  Thus a
 * factor of around 10 in the speed up is because of using the shared memory in this example. Note, 
 * however, that shared memory can only be shared among the threads within a single block.
 * 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKSIZE 512 

// Define some GPU timing utilities. These are invoked from the host program. Usage:
//     START_GPU;
//         kernelFunction <<< numBlocks, threadsPerBlock >>> (args)
//     STOP_GPU;
//     PRINT_GPU
// in the host code. This estimates the time for the kernel kernelFunction to run on the GPU.
// For a more extensive discusion, see Section 5.1.2 of the CUDA Best Practices Guide at
// http://developer.download.nvidia.com/compute/DevZone/docs/html/C/doc/CUDA_C_Best_Practices_Guide.pdf

float timeGPU;
hipEvent_t start, stop;
#define START_GPU hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
#define STOP_GPU hipEventRecord(stop, 0); hipEventSynchronize(stop);\
   hipEventElapsedTime(&timeGPU, start, stop);\
   hipEventDestroy(start);hipEventDestroy(stop);
#define PRINT_GPU printf("\n\nTime to compute on GPU: %f ms \n", timeGPU);

// Define a utility to check for CUDA errors.  Place it immediately after a CUDA kernel
// call in the host code. The initial cudaDeviceSynchronize() command ensures that the device
// has completed all preceding requested tasks.

#define CUDA_ERROR_CHECK hipDeviceSynchronize(); hipError_t error = hipGetLastError();\
   if(error != hipSuccess){printf("***CUDA error: %s\n", hipGetErrorString(error)); exit(-1);}\
   else{printf("\nNo CUDA errors detected\n" );}
   

// Device code.  Examines the elements of the array Array and puts the max value in MaxVal

__global__ void MaxValKernel(float* Array, float* MaxVal, int arraySize)
{
	__device__ float reductionMaxVal(int, float *);

	*MaxVal = reductionMaxVal(arraySize, Array);
}

/*
  Function to do parallel reduction maxVal. This should scale as ln N.  The parallel butterfly
  algorithm taken from the literature works generally.  My homegrown parallel version
  works as written for even number of entries in the array, so this algorithm can be used 
  for an odd number by padding the array with an added zero entry. Note that this version 
  assumes that all maxVals are within one block, so a max of 512 threads on 1.1 devices
  (presently blocksize is set to 512).  One option for larger maxVals is to break the array up 
  onto multiple blocks, use this algorithm on each block to get a block maxVal, and then maxVal
  the block maxVals.
*/

__device__ float reductionMaxVal(int length, float *array)
{
	float maxVal;
	
	// = 0 or 1 for parallel with shared memory, 2 for serial with global, 3 for serial with shared
	int maxValControl = 0;
	
	// Copy the array to be maxValed into shared memory and initialize
	__shared__ float sarray[BLOCKSIZE];
	int i = threadIdx.x;
	sarray[i] = 0.0f;
	if(i<length) sarray[i] = array[i];
	__syncthreads();
	
	
	if(maxValControl == 0)
	{
		// Parallel butterfly maxVal
		// see http://cs.anu.edu.au/files/systems/GPUWksp/PDFs/02_CUDAParallelProgrammingModel.pdf
		
		for(int bit=BLOCKSIZE/2; bit>0; bit /= 2)
		{
			if(i<length)
			{
				float t = sarray[i];
				if(sarray[i^bit] > t) t = sarray[i^bit];
				__syncthreads();
				sarray[i] = t;
				__syncthreads();
			}
		}
		
		maxVal = sarray[0];   // Contains the max value
	
	}
	else if(maxValControl == 1)
	{
		// Another home-made parallel version of a reduction maxVal. As written, this requires an even
		// number of entries in the array to be maxValed, so pad with a zero to handle odd number
		// (or rewrite to deal with odd number).
		
		int L=length;
		int steps = 0;
		int inc = 1;
		float t = 0;
		
		while(L > 2 )
		{
			steps ++;
			if(i < (length-inc))
				t = sarray[i];
				if(sarray[i+inc] > t) t = sarray[i+inc];
				__syncthreads();
				sarray[i] = t;
				__syncthreads();
			inc *= 2;
			L /= 2;
		}
		
		maxVal = sarray[0];   // This contains the maxVal
	}
	else if(maxValControl == 2)
	{	
		// For reference: Serial version of maximum finder running on the GPU and accessing global 
		// (not shared) memory.
	
		maxVal = 1.0e-20;
		for(int i=0; i<length; i++)
		{
			if(array[i] > maxVal)
			{
				maxVal = array[i];
			}
		}
	}
	else if(maxValControl == 3)
	{
		// For reference: Serial version of maximum finder running on the GPU and accessing shared 
		// memory.
	
		maxVal = 1.0e-20;
		for(int i=0; i<length; i++)
		{
			if(sarray[i] > maxVal)
			{
				maxVal = sarray[i];
			}
		}
	}
	
	return maxVal;
}



// Host code running on CPU
int main(void)
{
	int arraySize = 512;
	float* Array;
	float MaxVal;
	
	Array = (float*) malloc(sizeof(float) * arraySize);
	
	// Fill array with some numbers.  Make it a monotonic function peaked at
	// i = 300.  Then add some spikes to it so there are multiple local
	// maxima.
	
	for(int i=0; i<arraySize; i++)
	{
		Array[i] = 20.0f*expf(-(float) (i-300.0f)*(i-300.0f)/(float)200.0f);
	}
	
	// Add some spikes so that max is at i=100, corresponding to a value 42.6.
	Array[100] = 42.6f;
	Array[150] = 35.0f;
	Array[400] = 26.8f;
	Array[450] = 30.0f;
	
	// Set up device pointers
	float *devPtrArray;
	float* devPtrMaxVal; 
	
	// Allocate device memory
	hipMalloc((void**)&devPtrMaxVal, sizeof(float));
	hipMalloc((void**)&devPtrArray, arraySize * sizeof(float));
	
	// Copy array to device
	hipMemcpy(devPtrArray, Array, arraySize * sizeof(float), hipMemcpyHostToDevice);
	
	// Launch the kernel.
	
	START_GPU;     // Start timer for device code
	
	MaxValKernel<<<1, 512>>>(devPtrArray, devPtrMaxVal, arraySize);
	
	STOP_GPU;           // Stop timer for device code
	PRINT_GPU;          // Print timing for device code
	CUDA_ERROR_CHECK
	
	// Copy the maxVal back from the GPU to the host
	
	hipMemcpy(&MaxVal, devPtrMaxVal, sizeof(float), hipMemcpyDeviceToHost);
	
	// Print out all the values in the array
	for(int i=0; i< arraySize; i++)
	{
		printf("%d %8.3e\n", i, Array[i]);
	}
	
	printf("\nTime to compute on GPU: %f ms \n", timeGPU);
	
	printf("\nMax value in array (GPU) = %6.2f\n\n", MaxVal);
	 
    // Free the memory allocated on the device   
	hipFree(devPtrMaxVal);
	hipFree(devPtrArray);
	
	// Free the memory allocated on the CPU
	free(Array);
	
    return 0;
	
}
