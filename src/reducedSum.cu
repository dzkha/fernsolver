/* 
 * Example of using reducing (tree) type algorithms to parallelize finding the sum of
 * a set of numbers. On a GF 8600 GT the two parallel algorithms (sumControl = 0 or 1) 
 * are about 35 times faster than the serial algorithm also running on the GPU but using 
 * global memory (sumControl=2), for an array of 512 floats.  This is both because
 * the parallel algorithms scale as ln N while the serial algorithm scales as N, and because the
 * parallel algorithms use the shared memory while the serial one uses the (generally slower)
 * global memory in these tests. If we do the serial algorithm on the same computer but use
 * shared memory (set sumControl=3), the parallel algorithms are only 3.5 times faster.  Thus a
 * factor of 10 in the speed up is because of using the shared memory in this example. Note, 
 * however, that shared memory can only be shared among the threads within a single block.
 * 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKSIZE 512 

// Define some GPU timing utilities. These are invoked from the host program. Usage:
//     START_GPU;
//         kernelFunction <<< numBlocks, threadsPerBlock >>> (args)
//     STOP_GPU;
//     PRINT_GPU
// in the host code. This estimates the time for the kernel kernelFunction to run on the GPU.
// For a more extensive discusion, see Section 5.1.2 of the CUDA Best Practices Guide at
// http://developer.download.nvidia.com/compute/DevZone/docs/html/C/doc/CUDA_C_Best_Practices_Guide.pdf

float timeGPU;
hipEvent_t start, stop;
#define START_GPU hipEventCreate(&start); hipEventCreate(&stop); hipEventRecord(start, 0);
#define STOP_GPU hipEventRecord(stop, 0); hipEventSynchronize(stop);\
   hipEventElapsedTime(&timeGPU, start, stop);\
   hipEventDestroy(start);hipEventDestroy(stop);
#define PRINT_GPU printf("\n\nTime to compute on GPU: %f ms \n", timeGPU);

// Define a utility to check for CUDA errors.  Place it immediately after a CUDA kernel
// call in the host code. The initial cudaDeviceSynchronize() command ensures that the device
// has completed all preceding requested tasks.

#define CUDA_ERROR_CHECK hipDeviceSynchronize(); hipError_t error = hipGetLastError();\
   if(error != hipSuccess){printf("***CUDA error: %s\n", hipGetErrorString(error)); exit(-1);}\
   else{printf("\nNo CUDA errors detected\n" );}
   

// Device code.  Sums the elements of the array Array and puts the result in Sum

__global__ void SumKernel(float* Array, float* Sum, int arraySize)
{
	__device__ float reductionSum(int, float *);

	*Sum = reductionSum(arraySize, Array);
}

/*
  Function to do parallel reduction sum. This should scale as ln N.  The parallel butterfly
  algorithm taken from the literature works generally.  My homegrown parallel version
  works as written for even number of entries in the array, so this algorithm can be used 
  for an odd number by padding the array with an added zero entry. Note that this version 
  assumes that all summations are within one block, so a max of 512 threads on 1.1 devices
  (presently blocksize is set to 256).  One option for larger sums is to break the array up 
  onto multiple blocks, use this algorithm on each block to get a block sum, and then sum
  the block sums.
*/

__device__ float reductionSum(int length, float *array)
{
	float sum = 0.0f;
	
	// = 0 or 1 for parallel with shared memory, 2 for serial with global, 3 for serial with shared
	int sumControl = 0;
	
	// Copy the array to be summed into shared memory and initialize
	__shared__ float sarray[BLOCKSIZE];
	int i = threadIdx.x;
	sarray[i] = 0.0f;
	if(i<length) sarray[i] = array[i];
	__syncthreads();
	
	
	if(sumControl == 0)
	{
		// Parallel butterfly sum
		// see http://cs.anu.edu.au/files/systems/GPUWksp/PDFs/02_CUDAParallelProgrammingModel.pdf
		
		for(int bit=BLOCKSIZE/2; bit>0; bit /= 2)
		{
			if(i<length)
			{
				float t=sarray[i] + sarray[i^bit];
				__syncthreads();
				sarray[i] = t;
				__syncthreads();
			}
		}
		
		// The array entries sarray[i] with i<length/2 now all contain the sum
		
		sum = sarray[0];
	
	}
	else if(sumControl == 1)
	{
		// Another home-made parallel version of a reduction sum. As written, this requires an even
		// number of entries in the array to be summed, so pad with a zero to handle odd number
		// (or rewrite to deal with odd number).
		
		int L=length;
		int steps = 0;
		int inc = 1;
		float t = 0;
		
		while(L > 2 )
		{
			steps ++;
			if(i < (length-inc))
				t = sarray[i] + sarray[i+inc];
				__syncthreads();
				sarray[i] = t;
				__syncthreads();
			inc *= 2;
			L /= 2;
		}
		
		sum = sarray[0] + sarray[inc];  // This contains the sum
	}
	else if(sumControl == 2)
	{	
		// Serial version of sum accessing global (not shared) memory
	
		sum = 0.0f;
		for(int i=0; i<length; i++)
		{
			sum += array[i];
		}
	}
	else
	{
		// Serial version of sum accessing shared memory
	
		sum = 0.0f;
		for(int i=0; i<length; i++)
		{
			sum += sarray[i];
		}
	}
	
	return sum;
}



// Host code
int main(void)
{
	
	int arraySize = 512;
	float* Array;
	
	Array = (float*) malloc(sizeof(float) * arraySize);
	
	// Fill array with some numbers
	
	for(int i=0; i<arraySize; i++)
	{
		Array[i] = 2.0f * (float) i;
	}
	
	// Set up device pointers
	float *devPtrArray;
	
	float Sum;
	float* devPtrSum;                            
	hipMalloc((void**)&devPtrSum, sizeof(float));
	
	// Allocate device memory
	hipMalloc((void**)&devPtrArray, arraySize * sizeof(float));
	
	// Copy array to device
	hipMemcpy(devPtrArray, Array, arraySize * sizeof(float), hipMemcpyHostToDevice);
	
	// Launch the kernel.
	
	START_GPU;     // Start timer for device code
	
	SumKernel<<<1, 512>>>(devPtrArray, devPtrSum, arraySize);
	
	STOP_GPU;           // Stop timer for device code
	PRINT_GPU;          // Print timing for device code
	CUDA_ERROR_CHECK
	
	// Copy the sum back from the GPU to the host
	
	hipMemcpy(&Sum, devPtrSum, sizeof(float), hipMemcpyDeviceToHost);
	
	printf("\nSum array elements multiplied by 2 (GPU) = %6.2f\n\n", Sum);
	 
    // Free the memory allocated on the device   
	hipFree(devPtrSum);
	hipFree(devPtrArray);
	
	// Free the memory allocated on the CPU
	free(Array);
	
    return 0;
	
}

