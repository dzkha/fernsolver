#include "hip/hip_runtime.h"

#include <explicitgpuDouble.cuh>


void launchTestReduce()
{
	unsigned short a_length = 1024;
	size_t a_size = sizeof(double) * a_length;
	double *a = (double *) malloc(a_size);
	
	for (int i = 0; i < a_length; i++)
		a[i] = (double) i;
	
	double *devA;
	hipMalloc(&devA, a_size);
	hipMemcpy(devA, a, a_size, hipMemcpyHostToDevice);
	
	dim3 blocks(1);
	dim3 threads(512);
	testReduce<<<blocks, threads, a_size>>>(devA, a_length);
	CUDA_ERROR_CHECK
	
	hipMemcpy(a, devA, a_size, hipMemcpyDeviceToHost);
	printf("Sum: %f\n", a[0]);
}


__global__ void testReduce(double *a, unsigned short length)
{
	const int tid = threadIdx.x;
	double *as = (double *) dsmem;
	
	__syncthreads();
	
	// Perform reductions on the array
	unsigned int loops = 1u << 17;
	for (unsigned int loop = 0; loop < loops; loop++)
	{
		// Copy global array to shared
		for (int i = tid; i < length; i += blockDim.x)
			as[i] = a[i];
		
		__syncthreads();
		reduceSum(as, length);
		__syncthreads();
	}
	
	// Copy shared array to global
	for (int i = tid; i < length; i += blockDim.x)
		a[i] = as[i];
}


